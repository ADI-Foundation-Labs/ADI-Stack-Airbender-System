#include "hip/hip_runtime.h"
#include "arg_utils.cuh"
#include "context.cuh"
#include "ops_complex.cuh"
#include "vectorized.cuh"

using namespace field;
using namespace memory;

using bf = base_field;
using e2 = ext2_field;
using e4 = ext4_field;

// so I can use a u8 to represent 255 column indexes and 1 sentinel value
constexpr unsigned MAX_MEMORY_COLS = 256;
constexpr unsigned DOES_NOT_NEED_Z_OMEGA = UINT_MAX;

EXTERN __launch_bounds__(128, 8) __global__
    void deep_denom_at_z_kernel(vector_setter<e4, st_modifier::cs> denom_at_z, const e4 *z_ref, const unsigned log_n, const bool bit_reversed) {
  constexpr unsigned INV_BATCH = InvBatch<e4>::INV_BATCH;

  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n)
    return;

  const auto grid_size = unsigned(blockDim.x * gridDim.x);

  e4 per_elem_factor_invs[INV_BATCH];

  const e4 z = *z_ref;
  unsigned runtime_batch_size = 0;
  const unsigned log_shift = CIRCLE_GROUP_LOG_ORDER - log_n;
#pragma unroll
  for (unsigned i{0}, g{gid}; i < INV_BATCH; i++, g += grid_size)
    if (g < n) {
      const unsigned k = (bit_reversed ? __brev(g) >> (32 - log_n) : g) << log_shift;
      const auto x = get_power_of_w(k, false);
      per_elem_factor_invs[i] = e4::sub(x, z);
      runtime_batch_size++;
    }

  e4 per_elem_factors[INV_BATCH];

  if (runtime_batch_size < INV_BATCH) {
    batch_inv_registers<e4, INV_BATCH, false>(per_elem_factor_invs, per_elem_factors, runtime_batch_size);
  } else {
    batch_inv_registers<e4, INV_BATCH, true>(per_elem_factor_invs, per_elem_factors, runtime_batch_size);
  }

#pragma unroll
  for (unsigned i{0}, g{gid}; i < INV_BATCH; i++, g += grid_size)
    if (g < n)
      denom_at_z.set(g, per_elem_factors[i]);
}

extern "C" struct ColIdxsToChallengeIdxsMap {
  const unsigned map[MAX_MEMORY_COLS];
};

extern "C" struct ChallengesTimesEvalsSums {
  const e4 at_z_sum_neg;
  const e4 at_z_omega_sum_neg;
};

EXTERN __launch_bounds__(512, 2) __global__
    void deep_quotient_kernel(matrix_getter<bf, ld_modifier::cs> setup_cols, matrix_getter<bf, ld_modifier::cs> witness_cols,
                              matrix_getter<bf, ld_modifier::cs> memory_cols, matrix_getter<bf, ld_modifier::cs> stage_2_bf_cols,
                              vectorized_e4_matrix_getter<ld_modifier::cs> stage_2_e4_cols, vectorized_e4_matrix_getter<ld_modifier::cs> composition_col,
                              vector_getter<e4, ld_modifier::ca> denom_at_z, vector_getter<e4, ld_modifier::ca> setup_challenges_at_z,
                              vector_getter<e4, ld_modifier::ca> witness_challenges_at_z, vector_getter<e4, ld_modifier::ca> memory_challenges_at_z,
                              vector_getter<e4, ld_modifier::ca> stage_2_bf_challenges_at_z, vector_getter<e4, ld_modifier::ca> stage_2_e4_challenges_at_z,
                              vector_getter<e4, ld_modifier::ca> composition_challenge_at_z,
                              __grid_constant__ const StateLinkageConstraints state_linkage_constraints,
                              __grid_constant__ const ColIdxsToChallengeIdxsMap memory_cols_to_challenges_at_z_omega_map,
                              vector_getter<e4, ld_modifier::ca> witness_challenges_at_z_omega, vector_getter<e4, ld_modifier::ca> memory_challenges_at_z_omega,
                              vector_getter<e4, ld_modifier::ca> grand_product_challenge_at_z_omega,
                              const ChallengesTimesEvalsSums *challenges_times_evals_sums_ref, vectorized_e4_matrix_setter<st_modifier::cs> quotient,
                              const unsigned num_setup_cols, const unsigned num_witness_cols, const unsigned num_memory_cols,
                              const unsigned num_stage_2_bf_cols, const unsigned num_stage_2_e4_cols, const unsigned stage_2_memory_grand_product_offset,
                              const unsigned log_n, const bool bit_reversed) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n)
    return;

  setup_cols.add_row(gid);
  witness_cols.add_row(gid);
  memory_cols.add_row(gid);
  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);
  composition_col.add_row(gid);
  quotient.add_row(gid);

  e4 acc_z = e4::zero();
  e4 acc_z_omega = e4::zero();

  // Setup terms at z
  for (unsigned i = 0; i < num_setup_cols; i++) {
    const bf val = setup_cols.get_at_col(i);
    const e4 challenge = setup_challenges_at_z.get(i);
    acc_z = e4::add(acc_z, e4::mul(challenge, val));
  }

  // Witness terms at z
  for (unsigned i = 0; i < num_witness_cols; i++) {
    const bf val = witness_cols.get_at_col(i);
    const e4 challenge = witness_challenges_at_z.get(i);
    acc_z = e4::add(acc_z, e4::mul(challenge, val));
  }

  // Witness terms at z * omega (state linkage). Redundant loads, but negligible.
  for (unsigned i = 0; i < state_linkage_constraints.num_constraints; i++) {
    const bf val = witness_cols.get_at_col(state_linkage_constraints.dsts[i]);
    const e4 challenge = witness_challenges_at_z_omega.get(i);
    acc_z_omega = e4::add(acc_z_omega, e4::mul(challenge, val));
  }

  // Memory terms at z and z * omega
  {
    unsigned challenge_at_z_omega_idx = 0;
    for (unsigned i = 0; i < num_memory_cols; i++) {
      const bf val = memory_cols.get_at_col(i);
      const e4 challenge = memory_challenges_at_z.get(i);
      acc_z = e4::add(acc_z, e4::mul(challenge, val));
      const unsigned maybe_challenge_at_z_omega_idx = memory_cols_to_challenges_at_z_omega_map.map[i];
      if (maybe_challenge_at_z_omega_idx != DOES_NOT_NEED_Z_OMEGA) {
        const e4 challenge = memory_challenges_at_z_omega.get(challenge_at_z_omega_idx++);
        acc_z_omega = e4::add(acc_z_omega, e4::mul(challenge, val));
      }
    }
  }

  // Stage 2 bf terms at z
  for (unsigned i = 0; i < num_stage_2_bf_cols; i++) {
    const bf val = stage_2_bf_cols.get_at_col(i);
    const e4 challenge = stage_2_bf_challenges_at_z.get(i);
    acc_z = e4::add(acc_z, e4::mul(challenge, val));
  }

  // Stage 2 e4 terms at z and z * omega
  for (unsigned i = 0; i < num_stage_2_e4_cols; i++) {
    const e4 val = stage_2_e4_cols.get_at_col(i);
    const e4 challenge = stage_2_e4_challenges_at_z.get(i);
    acc_z = e4::add(acc_z, e4::mul(challenge, val));
    if (i == stage_2_memory_grand_product_offset) {
      const e4 challenge = grand_product_challenge_at_z_omega.get(0);
      acc_z_omega = e4::add(acc_z_omega, e4::mul(challenge, val));
    }
  }

  // Composition term at z
  const e4 val = composition_col.get();
  const e4 challenge = composition_challenge_at_z.get(0);
  acc_z = e4::add(acc_z, e4::mul(challenge, val));

  const e4 denom_z = denom_at_z.get(gid);
  const unsigned raw_row = bit_reversed ? __brev(gid) >> (32 - log_n) : gid;
  const unsigned row_shift = n - 1;
  const unsigned raw_shifted_row = (raw_row + row_shift >= n) ? raw_row + row_shift - n : raw_row + row_shift;
  const unsigned shifted_row = bit_reversed ? __brev(raw_shifted_row) >> (32 - log_n) : raw_shifted_row;
  const e4 denom_z_omega = denom_at_z.get(shifted_row);

  acc_z = e4::add(acc_z, challenges_times_evals_sums_ref->at_z_sum_neg);
  acc_z_omega = e4::add(acc_z_omega, challenges_times_evals_sums_ref->at_z_omega_sum_neg);
  acc_z = e4::mul(acc_z, denom_z);
  acc_z_omega = e4::mul(acc_z_omega, denom_z_omega);

  quotient.set(e4::add(acc_z, acc_z_omega));
}
