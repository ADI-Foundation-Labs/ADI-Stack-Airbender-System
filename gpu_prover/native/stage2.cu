#include "hip/hip_runtime.h"
#include "arg_utils.cuh"
#include "ops_complex.cuh"
#include "vectorized.cuh"

using namespace ::airbender::arg_utils;
using namespace ::airbender::field;
using namespace ::airbender::memory;
using namespace ::airbender::ops_complex;
using namespace ::airbender::vectorized;

namespace airbender::stage2 {

using bf = base_field;
using e2 = ext2_field;
using e4 = ext4_field;

// ENTRY_WIDTH = 1 logic is special-cased for range check lookups.
template <typename T, unsigned ENTRY_WIDTH>
DEVICE_FORCEINLINE void
aggregated_entry_invs_and_multiplicities_arg_impl(const T *challenges_ptr, matrix_getter<bf, ld_modifier::cs> witness_cols,
                                                  matrix_getter<bf, ld_modifier::cs> setup_cols, vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                                  // st_modifier::cg to cache stores for upcoming lookup_a_args_kernel
                                                  vector_setter<e4, st_modifier::cg> aggregated_entry_invs, const unsigned start_col_in_setup,
                                                  const unsigned multiplicities_src_cols_start, const unsigned multiplicities_dst_cols_start,
                                                  const unsigned num_multiplicities_cols, const unsigned num_table_rows_tail, const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by generic lookup args kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  stage_2_e4_cols.add_col(multiplicities_dst_cols_start);
  witness_cols.add_row(gid);
  witness_cols.add_col(multiplicities_src_cols_start);
  aggregated_entry_invs += gid;

  // for width = 1 (range check) the value is the row index. We can just use gid instead of reading from setup.
  if (ENTRY_WIDTH > 1) {
    setup_cols.add_row(gid);
    setup_cols.add_col(start_col_in_setup);
  }

  const auto [linearization_challenges, gamma] = *challenges_ptr;
  for (unsigned i = 0; i < num_multiplicities_cols; i++) {
    if (i == num_multiplicities_cols - 1 && gid >= num_table_rows_tail) {
      stage_2_e4_cols.set(e4::zero());
      return;
    }

    // for range checks, we can just use gid
    bf val;
    if (ENTRY_WIDTH == 1) {
      val = bf{gid};
    } else {
      val = setup_cols.get();
      setup_cols.add_col(1);
    }
    e4 denom = e4::add(gamma, val);
    if (ENTRY_WIDTH > 1) { // hint to compiler to optimize this part out if possible
#pragma unroll
      for (unsigned j = 1; j < ENTRY_WIDTH; j++) {
        const auto val = setup_cols.get();
        setup_cols.add_col(1);
        denom = e4::add(denom, e4::mul(linearization_challenges[j - 1], val));
      }
    }

    const e4 denom_inv{e4::inv(denom)};

    const auto multiplicity = witness_cols.get();
    stage_2_e4_cols.set(e4::mul(denom_inv, multiplicity));
    aggregated_entry_invs.set(denom_inv);

    witness_cols.add_col(1);
    aggregated_entry_invs += n - 1; // next iteration's warp accesses will be unaligned, but this is likely negligible overall
    stage_2_e4_cols.add_col(1);
  }
}

EXTERN __launch_bounds__(128, 8) __global__ void ab_range_check_aggregated_entry_invs_and_multiplicities_arg_kernel(
    const LookupChallenges *challenges, matrix_getter<bf, ld_modifier::cs> witness_cols, matrix_getter<bf, ld_modifier::cs> setup_cols,
    vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
    // st_modifier::cg because these will be reused by later kernels
    vector_setter<e4, st_modifier::cg> aggregated_entry_invs, const unsigned start_col_in_setup, const unsigned multiplicities_src_cols_start,
    const unsigned multiplicities_dst_cols_start, const unsigned num_multiplicities_cols, const unsigned num_table_rows_tail, const unsigned log_n) {
  aggregated_entry_invs_and_multiplicities_arg_impl<LookupChallenges, 1>(
      challenges, witness_cols, setup_cols, stage_2_e4_cols, aggregated_entry_invs, start_col_in_setup, multiplicities_src_cols_start,
      multiplicities_dst_cols_start, num_multiplicities_cols, num_table_rows_tail, log_n);
}

EXTERN __launch_bounds__(128, 8) __global__ void ab_decoder_aggregated_entry_invs_and_multiplicities_arg_kernel(
    const DecoderTableChallenges *challenges, matrix_getter<bf, ld_modifier::cs> witness_cols, matrix_getter<bf, ld_modifier::cs> setup_cols,
    vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
    // st_modifier::cg because these will be reused by later kernels
    vector_setter<e4, st_modifier::cg> aggregated_entry_invs, const unsigned start_col_in_setup, const unsigned multiplicities_src_cols_start,
    const unsigned multiplicities_dst_cols_start, const unsigned num_multiplicities_cols, const unsigned num_table_rows_tail, const unsigned log_n) {
  aggregated_entry_invs_and_multiplicities_arg_impl<DecoderTableChallenges, EXECUTOR_FAMILY_CIRCUIT_DECODER_TABLE_WIDTH>(
      challenges, witness_cols, setup_cols, stage_2_e4_cols, aggregated_entry_invs, start_col_in_setup, multiplicities_src_cols_start,
      multiplicities_dst_cols_start, num_multiplicities_cols, num_table_rows_tail, log_n);
}

EXTERN __launch_bounds__(128, 8) __global__ void ab_generic_aggregated_entry_invs_and_multiplicities_arg_kernel(
    const LookupChallenges *challenges, matrix_getter<bf, ld_modifier::cs> witness_cols, matrix_getter<bf, ld_modifier::cs> setup_cols,
    vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
    // st_modifier::cg because these will be reused by later kernels
    vector_setter<e4, st_modifier::cg> aggregated_entry_invs, const unsigned start_col_in_setup, const unsigned multiplicities_src_cols_start,
    const unsigned multiplicities_dst_cols_start, const unsigned num_multiplicities_cols, const unsigned num_table_rows_tail, const unsigned log_n) {
  aggregated_entry_invs_and_multiplicities_arg_impl<LookupChallenges, NUM_LOOKUP_ARGUMENT_KEY_PARTS>(
      challenges, witness_cols, setup_cols, stage_2_e4_cols, aggregated_entry_invs, start_col_in_setup, multiplicities_src_cols_start,
      multiplicities_dst_cols_start, num_multiplicities_cols, num_table_rows_tail, log_n);
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_handle_delegation_requests_kernel(__grid_constant__ const DelegationChallenges challenges,
                                              __grid_constant__ const DelegationRequestMetadata request_metadata,
                                              matrix_getter<bf, ld_modifier::cs> memory_cols,
                                              matrix_getter<bf, ld_modifier::cs> setup_cols,
                                              vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                              const unsigned delegation_aux_poly_col,
                                              const bool is_unrolled,
                                              const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  memory_cols.add_row(gid);

  const bf num = memory_cols.get_at_col(request_metadata.multiplicity_col);

  bf timestamp_low{};
  bf timestamp_high{};
  if (is_unrolled) {
    timestamp_low = memory_cols.get_at_col(request_metadata.timestamp_col);
    timestamp_low = bf::add(timestamp_low, request_metadata.in_cycle_write_idx);

    timestamp_high = memory_cols.get_at_col(request_metadata.timestamp_col + 1);
  } else {
    setup_cols.add_row(gid);

    timestamp_low = setup_cols.get_at_col(request_metadata.timestamp_col);
    timestamp_low = bf::add(timestamp_low, request_metadata.in_cycle_write_idx);

    timestamp_high = setup_cols.get_at_col(request_metadata.timestamp_col + 1);
    timestamp_high = bf::add(timestamp_high, request_metadata.memory_timestamp_high_from_circuit_idx);
  }

  e4 denom = challenges.gamma;
  denom = e4::add(denom, memory_cols.get_at_col(request_metadata.delegation_type_col));
  denom = e4::add(denom, e4::mul(challenges.linearization_challenges[0], memory_cols.get_at_col(request_metadata.abi_mem_offset_high_col)));
  denom = e4::add(denom, e4::mul(challenges.linearization_challenges[1], timestamp_low));
  denom = e4::add(denom, e4::mul(challenges.linearization_challenges[2], timestamp_high));

  const e4 denom_inv{e4::inv(denom)};
  stage_2_e4_cols.set_at_col(delegation_aux_poly_col, e4::mul(num, denom_inv));
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_process_delegations_kernel(__grid_constant__ const DelegationChallenges challenges,
                                       __grid_constant__ const DelegationProcessingMetadata processing_metadata,
                                       matrix_getter<bf, ld_modifier::cs> memory_cols,
                                       vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                       const unsigned delegation_aux_poly_col,
                                       const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  memory_cols.add_row(gid);

  const bf num = memory_cols.get_at_col(processing_metadata.multiplicity_col);

  e4 denom = challenges.gamma;
  denom = e4::add(denom, processing_metadata.delegation_type);
  denom = e4::add(denom, e4::mul(challenges.linearization_challenges[0], memory_cols.get_at_col(processing_metadata.abi_mem_offset_high_col)));
  denom = e4::add(denom, e4::mul(challenges.linearization_challenges[1], memory_cols.get_at_col(processing_metadata.write_timestamp_col)));
  denom = e4::add(denom, e4::mul(challenges.linearization_challenges[2], memory_cols.get_at_col(processing_metadata.write_timestamp_col + 1)));

  const e4 denom_inv{e4::inv(denom)};
  stage_2_e4_cols.set_at_col(delegation_aux_poly_col, e4::mul(num, denom_inv));
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_range_check_16_trivial_checks_kernel(
                               __grid_constant__ const RangeCheckArgsLayout range_check_16_layout,
                               matrix_getter<bf, ld_modifier::cs> witness_cols,
                               vector_getter<e4, ld_modifier::ca> aggregated_entry_invs_for_range_check_16,
                               matrix_setter<bf, st_modifier::cs> stage_2_bf_cols,
                               vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                               const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n - 1)
    return;

  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);
  witness_cols.add_row(gid);

  for (unsigned i = 0; i < range_check_16_layout.num_dst_cols; i++) {
    const unsigned src = 2 * i + range_check_16_layout.src_cols_start;
    const bf val0 = bf::into_canonical(witness_cols.get_at_col(src));
    const bf val1 = bf::into_canonical(witness_cols.get_at_col(src + 1));
    const auto entry0 = aggregated_entry_invs_for_range_check_16.get(val0.limb);
    const auto entry1 = aggregated_entry_invs_for_range_check_16.get(val1.limb);
    const auto bf_arg = bf::mul(val0, val1);
    const auto e4_arg = e4::add(entry0, entry1);
    stage_2_bf_cols.set_at_col(range_check_16_layout.bf_args_start + i, bf_arg);
    stage_2_e4_cols.set_at_col(range_check_16_layout.e4_args_start + i, e4_arg);
  }
}


EXTERN __launch_bounds__(128, 8) __global__
    void ab_range_check_expressions_kernel(
                               __grid_constant__ const TEMPORARYFlattenedLookupExpressionsLayout expressions,
                               matrix_getter<bf, ld_modifier::cs> witness_cols,
                               matrix_getter<bf, ld_modifier::cs> memory_cols,
                               vector_getter<e4, ld_modifier::ca> aggregated_entry_invs,
                               matrix_setter<bf, st_modifier::cs> stage_2_bf_cols,
                               vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                               const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n - 1)
    return;

  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);
  witness_cols.add_row(gid);
  memory_cols.add_row(gid);

  for (unsigned i = 0, expression_idx = 0, flat_term_idx = 0; i < expressions.num_expression_pairs; i++) {
    bf a_and_b[2];
    eval_a_and_b<true>(a_and_b, expressions, expression_idx, flat_term_idx, witness_cols, memory_cols, expressions.constant_terms_are_zero);
    a_and_b[0] = bf::into_canonical(a_and_b[0]);
    a_and_b[1] = bf::into_canonical(a_and_b[1]);
    const e4 entry_a = aggregated_entry_invs.get(a_and_b[0].limb);
    const e4 entry_b = aggregated_entry_invs.get(a_and_b[1].limb);
    const bf bf_arg = bf::mul(a_and_b[0], a_and_b[1]);
    const e4 e4_arg = e4::add(entry_a, entry_b);
    stage_2_bf_cols.set_at_col(expressions.bf_dst_cols[i], bf_arg);
    stage_2_e4_cols.set_at_col(expressions.e4_dst_cols[i], e4_arg);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_range_check_expressions_for_shuffle_ram_kernel(
                               __grid_constant__ const FlattenedLookupExpressionsForShuffleRamLayout expressions_for_shuffle_ram,
                               matrix_getter<bf, ld_modifier::cs> setup_cols,
                               matrix_getter<bf, ld_modifier::cs> witness_cols,
                               matrix_getter<bf, ld_modifier::cs> memory_cols,
                               vector_getter<e4, ld_modifier::ca> aggregated_entry_invs,
                               matrix_setter<bf, st_modifier::cs> stage_2_bf_cols,
                               vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                               const bf memory_timestamp_high_from_circuit_idx,
                               const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n - 1)
    return;

  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);
  setup_cols.add_row(gid);
  witness_cols.add_row(gid);
  memory_cols.add_row(gid);

  for (unsigned i = 0, expression_idx = 0, flat_term_idx = 0; i < expressions_for_shuffle_ram.num_expression_pairs; i++) {
    bf a_and_b[2];
    eval_a_and_b<true>(a_and_b, expressions_for_shuffle_ram, expression_idx, flat_term_idx, setup_cols, witness_cols, memory_cols);
    a_and_b[1] = bf::sub(a_and_b[1], memory_timestamp_high_from_circuit_idx);
    a_and_b[0] = bf::into_canonical(a_and_b[0]);
    a_and_b[1] = bf::into_canonical(a_and_b[1]);
    const e4 entry_a = aggregated_entry_invs.get(a_and_b[0].limb);
    const e4 entry_b = aggregated_entry_invs.get(a_and_b[1].limb);
    const bf bf_arg = bf::mul(a_and_b[0], a_and_b[1]);
    const e4 e4_arg = e4::add(entry_a, entry_b);
    stage_2_bf_cols.set_at_col(expressions_for_shuffle_ram.bf_dst_cols[i], bf_arg);
    stage_2_e4_cols.set_at_col(expressions_for_shuffle_ram.e4_dst_cols[i], e4_arg);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_lazy_init_range_checks_kernel(
                               __grid_constant__ const LazyInitTeardownLayouts lazy_init_teardown_layouts,
                               matrix_getter<bf, ld_modifier::cs> memory_cols,
                               vector_getter<e4, ld_modifier::ca> aggregated_entry_invs_for_range_check_16,
                               matrix_setter<bf, st_modifier::cs> stage_2_bf_cols,
                               vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                               const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n - 1)
    return;

  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);
  memory_cols.add_row(gid);

  for (unsigned i = 0; i < lazy_init_teardown_layouts.num_init_teardown_sets; i++) {
    const auto &lazy_init_teardown_layout = lazy_init_teardown_layouts.layouts[i];
    const bf val0 = bf::into_canonical(memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start));
    const bf val1 = bf::into_canonical(memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start + 1));
    const auto entry0 = aggregated_entry_invs_for_range_check_16.get(val0.limb);
    const auto entry1 = aggregated_entry_invs_for_range_check_16.get(val1.limb);
    const auto bf_arg = bf::mul(val0, val1);
    const auto e4_arg = e4::add(entry0, entry1);
    stage_2_bf_cols.set_at_col(lazy_init_teardown_layout.bf_arg_col, bf_arg);
    stage_2_e4_cols.set_at_col(lazy_init_teardown_layout.e4_arg_col, e4_arg);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_decoder_lookup_intermediate_poly_kernel(
                               matrix_getter<bf, ld_modifier::cs> memory_cols,
                               vector_getter<e4, ld_modifier::ca> aggregated_entry_invs_for_decoder_lookups,
                               vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                               const unsigned decoder_lookup_arg_col,
                               const unsigned predicate_col,
                               const unsigned pc_start_col,
                               const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  memory_cols.add_row(gid);

  // witness gen probably ensures execute predicate is canonical, but being careful doesn't hurt
  const unsigned predicate = bf::into_canonical(memory_cols.get_at_col(predicate_col)).limb;
  if (predicate) {
    const unsigned pc_high = bf::into_canonical(memory_cols.get_at_col(pc_start_col)).limb;
    const unsigned pc_low = bf::into_canonical(memory_cols.get_at_col(pc_start_col + 1)).limb;
    const unsigned pc = (pc_high << 16) | pc_low;
    const unsigned decoder_table_row = pc >> 2;

    const e4 aggregated_entry_inv = aggregated_entry_invs_for_decoder_lookups.get(decoder_table_row);

    stage_2_e4_cols.set_at_col(decoder_lookup_arg_col, aggregated_entry_inv);
  } else {
    stage_2_e4_cols.set_at_col(decoder_lookup_arg_col, e4::zero());
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_generic_lookup_intermediate_polys_kernel(
                               matrix_getter<unsigned, ld_modifier::cs> generic_lookups_args_to_table_entries_map,
                               vector_getter<e4, ld_modifier::ca> aggregated_entry_invs_for_generic_lookups,
                               matrix_setter<bf, st_modifier::cs> stage_2_bf_cols,
                               vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                               const unsigned generic_args_start,
                               const unsigned num_generic_args,
                               const unsigned num_stage_2_bf_cols,
                               const unsigned num_stage_2_e4_cols,
                               const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n)
    return;

  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);

  // For bf cols, the final row is reserved for c0 = 0 adjustments.
  // Here we take the opportunity to zero the final row for all stage 2 arg cols.
  if (gid == n - 1) {
    for (unsigned i = 0; i < num_stage_2_bf_cols; i++)
      stage_2_bf_cols.set_at_col(i, bf::zero());
    for (unsigned i = 0; i < num_stage_2_e4_cols; i++)
      stage_2_e4_cols.set_at_col(i, e4::zero());
    return;
  }

  generic_lookups_args_to_table_entries_map.add_row(gid);

  for (unsigned i = 0; i < num_generic_args; i++) {
    const unsigned absolute_row_index = generic_lookups_args_to_table_entries_map.get_at_col(i);
    const e4 aggregated_entry_inv = aggregated_entry_invs_for_generic_lookups.get(absolute_row_index);
    stage_2_e4_cols.set_at_col(generic_args_start + i, aggregated_entry_inv);
  }
}

DEVICE_FORCEINLINE
void grand_product_lazy_init_contributions(const MemoryChallenges &challenges,
                                           const LazyInitTeardownLayouts &layouts,
                                           matrix_getter<bf, ld_modifier::cs> memory_cols,
                                           vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                           e4 &num_over_denom_acc) {
  for (unsigned i = 0; i < layouts.num_init_teardown_sets; i++) {
    const auto &layout = layouts.layouts[i];

    e4 numerator{challenges.gamma};
    const bf address_low = memory_cols.get_at_col(layout.init_address_start);
    numerator = e4::add(numerator, e4::mul(challenges.address_low_challenge, address_low));
    const bf address_high = memory_cols.get_at_col(layout.init_address_start + 1);
    numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));

    e4 denom{numerator};
    const bf value_low = memory_cols.get_at_col(layout.teardown_value_start);
    denom = e4::add(denom, e4::mul(challenges.value_low_challenge, value_low));
    const bf value_high = memory_cols.get_at_col(layout.teardown_value_start + 1);
    denom = e4::add(denom, e4::mul(challenges.value_high_challenge, value_high));
    const bf timestamp_low = memory_cols.get_at_col(layout.teardown_timestamp_start);
    denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, timestamp_low));
    const bf timestamp_high = memory_cols.get_at_col(layout.teardown_timestamp_start + 1);
    denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, timestamp_high));

    // flush result
    if (i == 0) {
      num_over_denom_acc = numerator;
    } else {
      num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
    }
    e4 denom_inv{e4::inv(denom)};
    num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
    stage_2_e4_cols.set_at_col(layouts.grand_product_contributions_start + i, num_over_denom_acc);
  }
}

DEVICE_FORCEINLINE
void grand_product_ram_access_contributions(const MemoryChallenges &challenges,
                                            const ShuffleRamAccesses &shuffle_ram_accesses,
                                            matrix_getter<bf, ld_modifier::cs> setup_cols,
                                            matrix_getter<bf, ld_modifier::cs> memory_cols,
                                            vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                            const bf memory_timestamp_high_from_circuit_idx,
                                            const unsigned memory_args_start, e4 &num_over_denom_acc) {
  // first, read a couple values common across accesses:
  const bf write_timestamp_in_setup_low = setup_cols.get_at_col(shuffle_ram_accesses.write_timestamp_in_setup_start);
  const bf write_timestamp_in_setup_high = setup_cols.get_at_col(shuffle_ram_accesses.write_timestamp_in_setup_start + 1);
#pragma unroll 1
  for (unsigned i = 0; i < shuffle_ram_accesses.num_accesses; i++) {
    const auto &access = shuffle_ram_accesses.accesses[i];

    e4 numerator{challenges.gamma};
    const bf address_low = memory_cols.get_at_col(access.address_start);
    numerator = e4::add(numerator, e4::mul(challenges.address_low_challenge, address_low));

    if (access.is_register_only) {
      numerator = e4::add(numerator, bf::one());
    } else {
      const bf address_high = memory_cols.get_at_col(access.address_start + 1);
      numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));
      numerator = e4::add(numerator, memory_cols.get_at_col(access.maybe_is_register_start));
      // TODO: It's possible address_high is always zero when memory_cols.get_at_col(access.maybe_is_register_start) is 1, which suggests:
      // const bf is_reg = memory_cols.get_at_col(access.maybe_is_register_start);
      // numerator = e4::add(numerator, memory_cols.get_at_col(access.maybe_is_register_start));
      // if (is_reg.limb) {
      //   const bf address_high = memory_cols.get_at_col(access.address_start + 1);
      //   numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));
      // }
    }

    e4 denom{};

    if (access.is_write) {
      denom = numerator;

      const bf read_value_low = memory_cols.get_at_col(access.read_value_start);
      denom = e4::add(denom, e4::mul(challenges.value_low_challenge, read_value_low));
      const bf read_value_high = memory_cols.get_at_col(access.read_value_start + 1);
      denom = e4::add(denom, e4::mul(challenges.value_high_challenge, read_value_high));

      const bf write_value_low = memory_cols.get_at_col(access.maybe_write_value_start);
      numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, write_value_low));
      const bf write_value_high = memory_cols.get_at_col(access.maybe_write_value_start + 1);
      numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, write_value_high));
    } else {
      const bf value_low = memory_cols.get_at_col(access.read_value_start);
      numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, value_low));
      const bf value_high = memory_cols.get_at_col(access.read_value_start + 1);
      numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, value_high));

      denom = numerator;
    }

    const bf read_timestamp_low = memory_cols.get_at_col(access.read_timestamp_start);
    denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, read_timestamp_low));
    const bf read_timestamp_high = memory_cols.get_at_col(access.read_timestamp_start + 1);
    denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, read_timestamp_high));

    const bf access_index{i};
    const bf write_timestamp_low = bf::add(write_timestamp_in_setup_low, access_index);
    numerator = e4::add(numerator, e4::mul(challenges.timestamp_low_challenge, write_timestamp_low));
    const bf write_timestamp_high = bf::add(write_timestamp_in_setup_high, memory_timestamp_high_from_circuit_idx);
    numerator = e4::add(numerator, e4::mul(challenges.timestamp_high_challenge, write_timestamp_high));

    // flush result
    num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
    e4 denom_inv{e4::inv(denom)};
    num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
    stage_2_e4_cols.set_at_col(memory_args_start + i, num_over_denom_acc);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_lazy_init_and_ram_access_kernel(__grid_constant__ const MemoryChallenges challenges,
                                           __grid_constant__ const ShuffleRamAccesses shuffle_ram_accesses,
                                           __grid_constant__ const LazyInitTeardownLayouts lazy_init_teardown_layouts,
                                           matrix_getter<bf, ld_modifier::cs> setup_cols,
                                           matrix_getter<bf, ld_modifier::cs> memory_cols,
                                           vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                           const bf memory_timestamp_high_from_circuit_idx,
                                           const unsigned memory_args_start, const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  setup_cols.add_row(gid);
  memory_cols.add_row(gid);

  e4 num_over_denom_acc{};

  grand_product_lazy_init_contributions(challenges, lazy_init_teardown_layouts, memory_cols, stage_2_e4_cols, num_over_denom_acc);

  grand_product_ram_access_contributions(challenges, shuffle_ram_accesses, setup_cols, memory_cols, stage_2_e4_cols,
                                         memory_timestamp_high_from_circuit_idx, memory_args_start, num_over_denom_acc);
}

DEVICE_FORCEINLINE
void grand_product_ram_access_contributions(const MemoryChallenges &challenges,
                                            const ShuffleRamAccesses &shuffle_ram_accesses,
                                            matrix_getter<bf, ld_modifier::cs> setup_cols,
                                            matrix_getter<bf, ld_modifier::cs> memory_cols,
                                            vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                            const unsigned memory_args_start,
                                            const bool num_over_denom_acc_is_initialized,
                                            e4 &num_over_denom_acc) {
}

DEVICE_FORCEINLINE
void grand_product_machine_state_contributions(

) {

}

// one kernel handles all cases, to avoid re-reading e4 column
EXTERN __launch_bounds__(128, 8) __global__
    void ab_unrolled_grand_product_contributions_kernel(__grid_constant__ const MemoryChallenges memory_challenges,
                                                        __grid_constant__ const MachineStateChallenges machine_state_challenges,
                                                        __grid_constant__ const ShuffleRamAccesses shuffle_ram_accesses,
                                                        __grid_constant__ const LazyInitTeardownLayouts lazy_init_teardown_layouts,
                                                        matrix_getter<bf, ld_modifier::cs> setup_cols,
                                                        matrix_getter<bf, ld_modifier::cs> memory_cols,
                                                        vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                                        const unsigned ram_access_args_start,
                                                        const unsigned machine_state_permutation_arg_col,
                                                        const unsigned mask_arg_col,
                                                        const unsigned execute_col,
                                                        const bool process_ram_access,
                                                        const bool process_machine_state_permutation,
                                                        const bool process_mask,
                                                        const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  setup_cols.add_row(gid);
  memory_cols.add_row(gid);

  e4 num_over_denom_acc{};
  bool num_over_denom_acc_is_initialized = false;

  if (lazy_init_teardown_layouts.process_shuffle_ram_init) {
    grand_product_lazy_init_contributions(memory_challenges, lazy_init_teardown_layouts, memory_cols, stage_2_e4_cols, num_over_denom_acc);
    num_over_denom_acc_is_initialized = true;
  }

  // if (process_ram_access) {
  //   grand_product_unrolled_ram_access_contributions();
  // }

  if (process_machine_state_permutation) {
    grand_product_machine_state_contributions();
  }

  // apply mask
  if (process_mask) {
    const unsigned execute = bf::into_canonical(memory_cols.get_at_col(execute_col)).limb;
    if (execute) {
      stage_2_e4_cols.set_at_col(mask_arg_col, num_over_denom_acc);
    } else {
      stage_2_e4_cols.set_at_col(mask_arg_col, e4::one());
    }
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void ab_register_and_indirect_memory_args_kernel(__grid_constant__ const MemoryChallenges challenges,
                                                     __grid_constant__ const RegisterAndIndirectAccesses register_and_indirect_accesses,
                                                     matrix_getter<bf, ld_modifier::cs> memory_cols,
                                                     vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols, const unsigned memory_args_start,
                                                     const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  stage_2_e4_cols.add_col(memory_args_start);
  memory_cols.add_row(gid);

  // Compute write_timestamp_contribution, common across accesses
  const bf write_timestamp_low = memory_cols.get_at_col(register_and_indirect_accesses.write_timestamp_col);
  const e4 write_timestamp_low_contribution = e4::mul(write_timestamp_low, challenges.timestamp_low_challenge);
  const bf write_timestamp_high = memory_cols.get_at_col(register_and_indirect_accesses.write_timestamp_col + 1);
  const e4 write_timestamp_high_contribution = e4::mul(write_timestamp_high, challenges.timestamp_high_challenge);
  const e4 write_timestamp_contribution = e4::add(write_timestamp_low_contribution, write_timestamp_high_contribution);

  e4 num_over_denom_acc{};
  unsigned flat_indirect_idx = 0;

#pragma unroll 1
  for (unsigned i = 0; i < register_and_indirect_accesses.num_register_accesses; i++) {
    unsigned base_low;
    unsigned base_high;
    // Register contribution
    {
      const auto &register_access = register_and_indirect_accesses.register_accesses[i];

      // TODO: this initial constant contribution could be precomputed and stashed
      e4 numerator = register_access.gamma_plus_one_plus_address_low_contribution;

      e4 denom{};

      if (register_access.is_write) {
        denom = numerator;

        const bf read_value_low = memory_cols.get_at_col(register_access.read_value_col);
        denom = e4::add(denom, e4::mul(challenges.value_low_challenge, read_value_low));
        base_low = bf::into_canonical(read_value_low).limb;
        const bf read_value_high = memory_cols.get_at_col(register_access.read_value_col + 1);
        denom = e4::add(denom, e4::mul(challenges.value_high_challenge, read_value_high));
        base_high = bf::into_canonical(read_value_high).limb;

        const bf write_value_low = memory_cols.get_at_col(register_access.maybe_write_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, write_value_low));
        const bf write_value_high = memory_cols.get_at_col(register_access.maybe_write_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, write_value_high));
      } else {
        const bf value_low = memory_cols.get_at_col(register_access.read_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, value_low));
        base_low = bf::into_canonical(value_low).limb;
        const bf value_high = memory_cols.get_at_col(register_access.read_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, value_high));
        base_high = bf::into_canonical(value_high).limb;

        denom = numerator;
      }

      numerator = e4::add(numerator, write_timestamp_contribution);

      const bf read_timestamp_low = memory_cols.get_at_col(register_access.read_timestamp_col);
      denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, read_timestamp_low));
      const bf read_timestamp_high = memory_cols.get_at_col(register_access.read_timestamp_col + 1);
      denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, read_timestamp_high));

      if (i == 0)
        num_over_denom_acc = numerator;
      else
        num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
      e4 denom_inv{e4::inv(denom)};
      num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
      stage_2_e4_cols.set(num_over_denom_acc);
      stage_2_e4_cols.add_col(1);
    }

    const unsigned lim = flat_indirect_idx + register_and_indirect_accesses.indirect_accesses_per_register_access[i];
#pragma unroll 1
    for (; flat_indirect_idx < lim; flat_indirect_idx++) {
      const auto &indirect_access = register_and_indirect_accesses.indirect_accesses[flat_indirect_idx];

      // imitates prover/src/prover_stages/stage2_utils.rs
      unsigned address_low_u32 = base_low + indirect_access.offset_constant;
      const unsigned of_low_0 = address_low_u32 >> 16;
      address_low_u32 = address_low_u32 & 0x0000ffff;
      // account for variable_dependent offset, if used
      unsigned of_low_1 = 0;
      if (indirect_access.has_variable_dependent) {
        const bf v = memory_cols.get_at_col(indirect_access.maybe_variable_dependent_col);
        const bf v_canonical = bf::into_canonical(v);
        const unsigned extra_low = indirect_access.maybe_variable_dependent_coeff * v_canonical.limb;
        address_low_u32 = address_low_u32 + extra_low;
        of_low_1 = address_low_u32 >> 16;
        address_low_u32 = address_low_u32 & 0x0000ffff;
      }
      const bf address_low = bf{address_low_u32};
      // this should never overflow, because our address space should be representable with 32 bits.
      const bf address_high = bf{base_high + (of_low_0 | of_low_1)};

      e4 numerator{challenges.gamma};
      numerator = e4::add(numerator, e4::mul(challenges.address_low_challenge, address_low));
      numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));

      e4 denom{};

      if (indirect_access.has_write) {
        denom = numerator;

        const bf read_value_low = memory_cols.get_at_col(indirect_access.read_value_col);
        denom = e4::add(denom, e4::mul(challenges.value_low_challenge, read_value_low));
        const bf read_value_high = memory_cols.get_at_col(indirect_access.read_value_col + 1);
        denom = e4::add(denom, e4::mul(challenges.value_high_challenge, read_value_high));

        const bf write_value_low = memory_cols.get_at_col(indirect_access.maybe_write_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, write_value_low));
        const bf write_value_high = memory_cols.get_at_col(indirect_access.maybe_write_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, write_value_high));
      } else {
        const bf value_low = memory_cols.get_at_col(indirect_access.read_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, value_low));
        const bf value_high = memory_cols.get_at_col(indirect_access.read_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, value_high));

        denom = numerator;
      }

      numerator = e4::add(numerator, write_timestamp_contribution);

      const bf read_timestamp_low = memory_cols.get_at_col(indirect_access.read_timestamp_col);
      denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, read_timestamp_low));
      const bf read_timestamp_high = memory_cols.get_at_col(indirect_access.read_timestamp_col + 1);
      denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, read_timestamp_high));

      // flush result
      num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
      e4 denom_inv{e4::inv(denom)};
      num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
      stage_2_e4_cols.set(num_over_denom_acc);
      stage_2_e4_cols.add_col(1);
    }
  }
}

} // namespace airbender::stage2
