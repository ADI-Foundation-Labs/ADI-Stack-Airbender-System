#include "hip/hip_runtime.h"
#include "layout.cuh"
#include "memory.cuh"
#include "option.cuh"
#include "trace_main.cuh"

using namespace ::airbender::witness::layout;
using namespace ::airbender::witness::memory;
using namespace ::airbender::witness::option;
using namespace ::airbender::witness::trace::main;

namespace airbender::witness::memory::main {

struct MainMemorySubtree {
  const ShuffleRamInitAndTeardownLayouts shuffle_ram_init_and_teardown_layouts;
  const ShuffleRamAccessSets shuffle_ram_access_sets;
  const OptionU32::Option<DelegationRequestLayout> delegation_request_layout;
};

template <bool COMPUTE_WITNESS>
DEVICE_FORCEINLINE void process_shuffle_ram_access_sets(const ShuffleRamAccessSets &shuffle_ram_access_sets,
                                                        const MemoryQueriesTimestampComparisonAuxVars &memory_queries_timestamp_comparison_aux_vars,
                                                        const MainTrace &oracle, const TimestampScalar timestamp_high_from_circuit_sequence,
                                                        const matrix_setter<bf, st_modifier::cg> memory, const matrix_setter<bf, st_modifier::cg> witness,
                                                        const unsigned index) {
#pragma unroll
  for (u32 i = 0; i < MAX_SHUFFLE_RAM_ACCESS_SETS_COUNT; ++i) {
    if (i == shuffle_ram_access_sets.count)
      break;
    const auto [tag, payload] = shuffle_ram_access_sets.sets[i];
    ShuffleRamAddressEnum address = {};
    ColumnSet<NUM_TIMESTAMP_COLUMNS_FOR_RAM> read_timestamp_columns = {};
    ColumnSet<REGISTER_SIZE> read_value_columns = {};
    switch (tag) {
    case Readonly: {
      auto columns = payload.shuffle_ram_query_read_columns;
      address = columns.address;
      read_timestamp_columns = columns.read_timestamp;
      read_value_columns = columns.read_value;
      break;
    }
    case Write: {
      const auto columns = payload.shuffle_ram_query_write_columns;
      address = columns.address;
      read_timestamp_columns = columns.read_timestamp;
      read_value_columns = columns.read_value;
      break;
    }
    }
    switch (address.tag) {
    case RegisterOnly: {
      const auto register_index = address.payload.register_only_access_address.register_index;
      const u16 value = oracle.get_witness_from_placeholder<u16>({ShuffleRamAddress, i}, index);
      write_u16_value(register_index, value, memory);
      PRINT_U16(M, register_index, value);
      break;
    }
    case RegisterOrRam: {
      const auto [is_register_columns, address_columns] = address.payload.register_or_ram_access_address;
      const bool is_register_value = oracle.get_witness_from_placeholder<bool>({ShuffleRamIsRegisterAccess, i}, index);
      write_bool_value(is_register_columns, is_register_value, memory);
      PRINT_U16(M, is_register_columns, is_register_value);
      const u32 address_value = oracle.get_witness_from_placeholder<u32>({ShuffleRamAddress, i}, index);
      write_u32_value(address_columns, address_value, memory);
      PRINT_U32(M, address_columns, address_value);
      break;
    }
    }
    const TimestampData read_timestamp_value = oracle.get_witness_from_placeholder<TimestampData>({ShuffleRamReadTimestamp, i}, index);
    write_timestamp_value(read_timestamp_columns, read_timestamp_value, memory);
    PRINT_TS(M, read_timestamp_columns, read_timestamp_value);
    const u32 read_value_value = oracle.get_witness_from_placeholder<u32>({ShuffleRamReadValue, i}, index);
    write_u32_value(read_value_columns, read_value_value, memory);
    PRINT_U32(M, read_value_columns, read_value_value);
    if (tag == Write) {
      const auto write_value_columns = payload.shuffle_ram_query_write_columns.write_value;
      const u32 write_value_value = oracle.get_witness_from_placeholder<u32>({ShuffleRamWriteValue, i}, index);
      write_u32_value(write_value_columns, write_value_value, memory);
      PRINT_U32(M, write_value_columns, write_value_value);
    }
    if (!COMPUTE_WITNESS)
      continue;
    const TimestampScalar write_timestamp_base =
        timestamp_high_from_circuit_sequence + (static_cast<TimestampScalar>(index + 1) << TimestampData::NUM_EMPTY_BITS_FOR_RAM_TIMESTAMP);
    const ColumnAddress borrow_address = memory_queries_timestamp_comparison_aux_vars.addresses[i];
    const u32 read_timestamp_low = read_timestamp_value.get_low();
    const TimestampData write_timestamp = TimestampData::from_scalar(write_timestamp_base + i);
    const u32 write_timestamp_low = write_timestamp.get_low();
    const bool intermediate_borrow = TimestampData::sub_borrow(read_timestamp_low, write_timestamp_low).y;
    write_bool_value(borrow_address, intermediate_borrow, witness);
    PRINT_U16(W, borrow_address, intermediate_borrow);
  }
}

DEVICE_FORCEINLINE void process_delegation_requests(const DelegationRequestLayout &delegation_request_layout, const MainTrace &oracle,
                                                    const matrix_setter<bf, st_modifier::cg> memory, const unsigned index) {
  const auto [multiplicity, delegation_type, abi_mem_offset_high] = delegation_request_layout;
  const bool execute_delegation_value = oracle.get_witness_from_placeholder<bool>({ExecuteDelegation}, index);
  write_bool_value(multiplicity, execute_delegation_value, memory);
  PRINT_U16(M, multiplicity, execute_delegation_value);
  const u16 delegation_type_value = oracle.get_witness_from_placeholder<u16>({DelegationType}, index);
  write_u16_value(delegation_type, delegation_type_value, memory);
  PRINT_U16(M, delegation_type, delegation_type_value);
  const u16 abi_mem_offset_high_value = oracle.get_witness_from_placeholder<u16>({DelegationABIOffset}, index);
  write_u16_value(abi_mem_offset_high, abi_mem_offset_high_value, memory);
  PRINT_U16(M, abi_mem_offset_high, abi_mem_offset_high_value);
}

template <bool COMPUTE_WITNESS>
DEVICE_FORCEINLINE void generate(const MainMemorySubtree &subtree, const MemoryQueriesTimestampComparisonAuxVars &memory_queries_timestamp_comparison_aux_vars,
                                 const ShuffleRamInitsAndTeardowns &inits_and_teardowns, const ShuffleRamAuxComparisonSets &aux_comparison_sets,
                                 const MainTrace &oracle, const TimestampScalar timestamp_high_from_circuit_sequence,
                                 matrix_setter<bf, st_modifier::cg> memory, matrix_setter<bf, st_modifier::cg> witness, const unsigned count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  memory.add_row(gid);
  witness.add_row(gid);
  process_inits_and_teardowns<COMPUTE_WITNESS>(subtree.shuffle_ram_init_and_teardown_layouts, inits_and_teardowns, aux_comparison_sets, memory, witness, count,
                                               gid);
  process_shuffle_ram_access_sets<COMPUTE_WITNESS>(subtree.shuffle_ram_access_sets, memory_queries_timestamp_comparison_aux_vars, oracle,
                                                   timestamp_high_from_circuit_sequence, memory, witness, gid);
  if (subtree.delegation_request_layout.tag == OptionU32::Some)
    process_delegation_requests(subtree.delegation_request_layout.value, oracle, memory, gid);
}

EXTERN __global__ void ab_generate_memory_values_main_kernel(const __grid_constant__ MainMemorySubtree subtree,
                                                             const __grid_constant__ ShuffleRamInitsAndTeardowns inits_and_teardowns,
                                                             const __grid_constant__ MainTrace oracle, const matrix_setter<bf, st_modifier::cg> memory,
                                                             const unsigned count) {
  generate<false>(subtree, {}, inits_and_teardowns, {}, oracle, {}, memory, memory, count);
}

EXTERN __global__ void ab_generate_memory_and_witness_values_main_kernel(
    const __grid_constant__ MainMemorySubtree subtree,
    const __grid_constant__ MemoryQueriesTimestampComparisonAuxVars memory_queries_timestamp_comparison_aux_vars,
    const __grid_constant__ ShuffleRamInitsAndTeardowns inits_and_teardowns, const __grid_constant__ ShuffleRamAuxComparisonSets aux_comparison_sets,
    const __grid_constant__ MainTrace oracle, const __grid_constant__ TimestampScalar timestamp_high_from_circuit_sequence,
    const matrix_setter<bf, st_modifier::cg> memory, const matrix_setter<bf, st_modifier::cg> witness, const unsigned count) {
  generate<true>(subtree, memory_queries_timestamp_comparison_aux_vars, inits_and_teardowns, aux_comparison_sets, oracle, timestamp_high_from_circuit_sequence,
                 memory, witness, count);
}

} // namespace airbender::witness::memory::main