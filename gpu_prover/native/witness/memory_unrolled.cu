#include "hip/hip_runtime.h"
#include "layout.cuh"
#include "memory.cuh"
#include "option.cuh"
#include "placeholder.cuh"
#include "trace_unrolled.cuh"

using namespace ::airbender::witness::layout;
using namespace ::airbender::witness::memory;
using namespace ::airbender::witness::option;
using namespace ::airbender::witness::placeholder;
using namespace ::airbender::witness::trace::unrolled;

namespace airbender::witness::memory::unrolled {

struct UnrolledFamilyMemorySubtree {
  const OptionU32::Option<DelegationRequestLayout> delegation_request_layout;
  const MachineStatePermutationVariables machine_state_layout;
  const IntermediateStatePermutationVariables intermediate_state_layout;
  const ShuffleRamAccessSets shuffle_ram_access_sets;
};

template <bool COMPUTE_WITNESS, typename ORACLE>
DEVICE_FORCEINLINE void process_machine_state_assuming_preprocessed_decoder(
    const UnrolledFamilyMemorySubtree &subtree, const OptionU32::Option<ColumnAddress> &executor_family_circuit_next_timestamp_aux_var, const ORACLE &oracle,
    const matrix_setter<bf, st_modifier::cg> memory, const matrix_setter<bf, st_modifier::cg> witness, u32 *const __restrict__ decoder_lookup_mapping,
    const unsigned index) {
  const IntermediateStatePermutationVariables input_state_and_decoder_parts = subtree.intermediate_state_layout;
  const ColumnSet<1> execute_column = input_state_and_decoder_parts.execute;
  const bool execute_value = oracle.template get_witness_from_placeholder<bool>({ExecuteOpcodeFamilyCycle}, index);
  write_bool_value(execute_column, execute_value, memory);
  PRINT_U16(M, execute_column, execute_value);
  const ColumnSet<2> initial_pc_columns = input_state_and_decoder_parts.pc;
  const u32 initial_pc_value = oracle.template get_witness_from_placeholder<u32>({PcInit}, index);
  write_u32_value(initial_pc_columns, initial_pc_value, memory);
  PRINT_U32(M, initial_pc_columns, initial_pc_value);
  const ColumnSet<NUM_TIMESTAMP_COLUMNS_FOR_RAM> initial_timestamp_columns = input_state_and_decoder_parts.timestamp;
  const TimestampData initial_timestamp_value = oracle.template get_witness_from_placeholder<TimestampData>({OpcodeFamilyCycleInitialTimestamp}, index);
  write_timestamp_value(initial_timestamp_columns, initial_timestamp_value, memory);
  PRINT_TS(M, initial_timestamp_columns, initial_timestamp_value);
  const auto [pc_columns, final_ts_columns] = subtree.machine_state_layout;
  const u32 pc_value = oracle.template get_witness_from_placeholder<u32>({PcFin}, index);
  write_u32_value(pc_columns, pc_value, memory);
  PRINT_U32(M, pc_columns, pc_value);
  TimestampData final_ts_value = oracle.template get_witness_from_placeholder<TimestampData>({OpcodeFamilyCycleInitialTimestamp}, index);
  const bool intermediate_carry_value = final_ts_value.increment();
  write_timestamp_value(final_ts_columns, final_ts_value, memory);
  PRINT_TS(M, final_ts_columns, final_ts_value);
  const ExecutorFamilyDecoderData decoder_data = oracle.get_executor_family_data(index);
  if (input_state_and_decoder_parts.circuit_family_extra_mask.tag == MemorySubtree) {
    const u32 circuit_family_extra_mask = input_state_and_decoder_parts.circuit_family_extra_mask.offset;
    const auto family_mask_column = ColumnSet<1>{circuit_family_extra_mask, 1};
    const u8 family_mask_value = decoder_data.opcode_family_bits;
    write_u8_value(family_mask_column, family_mask_value, memory);
    PRINT_U8(M, family_mask_column, family_mask_value);
  }
  if (!COMPUTE_WITNESS)
    return;
  if (executor_family_circuit_next_timestamp_aux_var.tag == OptionU32::Some) {
    const ColumnAddress immediate_carry_column = executor_family_circuit_next_timestamp_aux_var.value;
    write_bool_value(immediate_carry_column, intermediate_carry_value, witness);
    PRINT_U16(W, immediate_carry_column, intermediate_carry_value);
  }
  if (input_state_and_decoder_parts.rs2_index.tag == WitnessSubtree) {
    const u32 offset = input_state_and_decoder_parts.rs2_index.offset;
    const auto rs2_index_column = ColumnSet<1>{offset, 1};
    const u8 rs2_index_value = decoder_data.rs2_index;
    write_u8_value(rs2_index_column, rs2_index_value, witness);
    PRINT_U8(W, rs2_index_column, rs2_index_value);
  }
  if (input_state_and_decoder_parts.rd_index.tag == WitnessSubtree) {
    const u32 offset = input_state_and_decoder_parts.rd_index.offset;
    const auto rd_index_column = ColumnSet<1>{offset, 1};
    const u8 rd_index_value = decoder_data.rd_index;
    write_u8_value(rd_index_column, rd_index_value, witness);
    PRINT_U8(W, rd_index_column, rd_index_value);
  }
  if (input_state_and_decoder_parts.circuit_family_extra_mask.tag == WitnessSubtree) {
    const u32 circuit_family_extra_mask = input_state_and_decoder_parts.circuit_family_extra_mask.offset;
    const auto family_mask_column = ColumnSet<1>{circuit_family_extra_mask, 1};
    const u8 family_mask_value = decoder_data.opcode_family_bits;
    write_u8_value(family_mask_column, family_mask_value, witness);
    PRINT_U8(W, family_mask_column, family_mask_value);
  }
  if (input_state_and_decoder_parts.decoder_witness_is_in_memory)
    return;
  const ColumnSet<1> rd_is_zero_column = input_state_and_decoder_parts.rd_is_zero;
  const bool rd_is_zero_value = decoder_data.rd_is_zero;
  write_bool_value(rd_is_zero_column, rd_is_zero_value, witness);
  PRINT_U16(W, rd_is_zero_column, rd_is_zero_value);
  const ColumnSet<REGISTER_SIZE> imm_columns = input_state_and_decoder_parts.imm;
  const u32 imm_value = decoder_data.imm;
  write_u32_value(imm_columns, imm_value, witness);
  PRINT_U32(W, imm_columns, imm_value);
  const ColumnSet<1> funct3_column = input_state_and_decoder_parts.funct3;
  const u8 funct3_value = decoder_data.funct3;
  write_u8_value(funct3_column, funct3_value, witness);
  PRINT_U8(W, funct3_column, funct3_value);
  decoder_lookup_mapping[index] = execute_value ? initial_pc_value / 4 : 0xffffffff;
}

template <bool COMPUTE_WITNESS, typename ORACLE>
DEVICE_FORCEINLINE void process_shuffle_ram_access_sets(const ShuffleRamAccessSets &shuffle_ram_access_sets,
                                                        const MemoryQueriesTimestampComparisonAuxVars &memory_queries_timestamp_comparison_aux_vars,
                                                        const ORACLE &oracle, const matrix_setter<bf, st_modifier::cg> memory,
                                                        const matrix_setter<bf, st_modifier::cg> witness, const unsigned index) {
  const TimestampScalar cycle_timestamp = oracle.template get_witness_from_placeholder<TimestampData>({OpcodeFamilyCycleInitialTimestamp}, index).as_scalar();
#pragma unroll
  for (u32 i = 0; i < MAX_SHUFFLE_RAM_ACCESS_SETS_COUNT; ++i) {
    if (i == shuffle_ram_access_sets.count)
      break;
    const auto [tag, payload] = shuffle_ram_access_sets.sets[i];
    ShuffleRamAddressEnum address = {};
    ColumnSet<NUM_TIMESTAMP_COLUMNS_FOR_RAM> read_timestamp_columns = {};
    ColumnSet<REGISTER_SIZE> read_value_columns = {};
    switch (tag) {
    case Readonly: {
      auto columns = payload.shuffle_ram_query_read_columns;
      address = columns.address;
      read_timestamp_columns = columns.read_timestamp;
      read_value_columns = columns.read_value;
      break;
    }
    case Write: {
      const auto columns = payload.shuffle_ram_query_write_columns;
      address = columns.address;
      read_timestamp_columns = columns.read_timestamp;
      read_value_columns = columns.read_value;
      break;
    }
    }
    switch (address.tag) {
    case RegisterOnly: {
      const auto register_index = address.payload.register_only_access_address.register_index;
      const u8 value = oracle.template get_witness_from_placeholder<u8>({ShuffleRamAddress, i}, index);
      write_u8_value(register_index, value, memory);
      PRINT_U8(M, register_index, value);
      break;
    }
    case RegisterOrRam: {
      const auto [is_register_columns, address_columns] = address.payload.register_or_ram_access_address;
      const bool is_register_value = oracle.template get_witness_from_placeholder<bool>({ShuffleRamIsRegisterAccess, i}, index);
      write_bool_value(is_register_columns, is_register_value, memory);
      PRINT_U16(M, is_register_columns, is_register_value);
      const u32 address_value = oracle.template get_witness_from_placeholder<u32>({ShuffleRamAddress, i}, index);
      write_u32_value(address_columns, address_value, memory);
      PRINT_U32(M, address_columns, address_value);
      break;
    }
    }
    const TimestampData read_timestamp_value = oracle.template get_witness_from_placeholder<TimestampData>({ShuffleRamReadTimestamp, i}, index);
    write_timestamp_value(read_timestamp_columns, read_timestamp_value, memory);
    PRINT_TS(M, read_timestamp_columns, read_timestamp_value);
    const u32 read_value_value = oracle.template get_witness_from_placeholder<u32>({ShuffleRamReadValue, i}, index);
    write_u32_value(read_value_columns, read_value_value, memory);
    PRINT_U32(M, read_value_columns, read_value_value);
    if (tag == Write) {
      const auto write_value_columns = payload.shuffle_ram_query_write_columns.write_value;
      const u32 write_value_value = oracle.template get_witness_from_placeholder<u32>({ShuffleRamWriteValue, i}, index);
      write_u32_value(write_value_columns, write_value_value, memory);
      PRINT_U32(M, write_value_columns, write_value_value);
    }
    if (!COMPUTE_WITNESS)
      continue;
    const ColumnAddress borrow_address = memory_queries_timestamp_comparison_aux_vars.addresses[i];
    const u32 read_timestamp_low = read_timestamp_value.get_low();
    const TimestampData write_timestamp = TimestampData::from_scalar(cycle_timestamp + i);
    const u32 write_timestamp_low = write_timestamp.get_low();
    const bool intermediate_borrow = TimestampData::sub_borrow(read_timestamp_low, write_timestamp_low).y;
    write_bool_value(borrow_address, intermediate_borrow, witness);
    PRINT_U16(W, borrow_address, intermediate_borrow);
  }
}

template <typename ORACLE>
DEVICE_FORCEINLINE void process_delegation_requests(const DelegationRequestLayout &delegation_request_layout, const ORACLE &oracle,
                                                    const matrix_setter<bf, st_modifier::cg> memory, const unsigned index) {
  const auto [multiplicity, delegation_type, abi_mem_offset_high] = delegation_request_layout;
  const bool execute_delegation_value = oracle.template get_witness_from_placeholder<bool>({ExecuteDelegation}, index);
  write_bool_value(multiplicity, execute_delegation_value, memory);
  PRINT_U16(M, multiplicity, execute_delegation_value);
  const u16 delegation_type_value = oracle.template get_witness_from_placeholder<u16>({DelegationType}, index);
  write_u16_value(delegation_type, delegation_type_value, memory);
  PRINT_U16(M, delegation_type, delegation_type_value);
  if (abi_mem_offset_high.num_elements == 0)
    return;
  const u16 abi_mem_offset_high_value = oracle.template get_witness_from_placeholder<u16>({DelegationABIOffset}, index);
  write_u16_value(abi_mem_offset_high, abi_mem_offset_high_value, memory);
  PRINT_U16(M, abi_mem_offset_high, abi_mem_offset_high_value);
}

template <bool COMPUTE_WITNESS, typename ORACLE>
DEVICE_FORCEINLINE void generate_family(const UnrolledFamilyMemorySubtree &subtree,
                                        const OptionU32::Option<ColumnAddress> &executor_family_circuit_next_timestamp_aux_var,
                                        const MemoryQueriesTimestampComparisonAuxVars &memory_queries_timestamp_comparison_aux_vars, const ORACLE &oracle,
                                        matrix_setter<bf, st_modifier::cg> memory, matrix_setter<bf, st_modifier::cg> witness,
                                        u32 *const __restrict__ decoder_lookup_mapping, const unsigned count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  memory.add_row(gid);
  witness.add_row(gid);
  process_machine_state_assuming_preprocessed_decoder<COMPUTE_WITNESS>(subtree, executor_family_circuit_next_timestamp_aux_var, oracle, memory, witness,
                                                                       decoder_lookup_mapping, gid);
  process_shuffle_ram_access_sets<COMPUTE_WITNESS>(subtree.shuffle_ram_access_sets, memory_queries_timestamp_comparison_aux_vars, oracle, memory, witness, gid);
  if (subtree.delegation_request_layout.tag == OptionU32::Some)
    process_delegation_requests(subtree.delegation_request_layout.value, oracle, memory, gid);
}

template <bool COMPUTE_WITNESS>
DEVICE_FORCEINLINE void generate_inits_and_teardowns(const ShuffleRamInitAndTeardownLayouts &init_and_teardown_layouts,
                                                     const ShuffleRamInitsAndTeardowns &inits_and_teardowns,
                                                     const ShuffleRamAuxComparisonSets &aux_comparison_sets, matrix_setter<bf, st_modifier::cg> memory,
                                                     matrix_setter<bf, st_modifier::cg> witness, const unsigned count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  memory.add_row(gid);
  witness.add_row(gid);
  process_inits_and_teardowns<COMPUTE_WITNESS>(init_and_teardown_layouts, inits_and_teardowns, aux_comparison_sets, memory, witness, count, gid);
}

EXTERN __global__ void ab_generate_memory_values_unrolled_memory_kernel(const __grid_constant__ UnrolledFamilyMemorySubtree subtree,
                                                                        const __grid_constant__ UnrolledMemoryOracle oracle,
                                                                        const matrix_setter<bf, st_modifier::cg> memory, const unsigned count) {
  generate_family<false>(subtree, {}, {}, oracle, memory, memory, nullptr, count);
}

EXTERN __global__ void ab_generate_memory_values_unrolled_non_memory_kernel(const __grid_constant__ UnrolledFamilyMemorySubtree subtree,
                                                                            const __grid_constant__ UnrolledNonMemoryOracle oracle,
                                                                            const matrix_setter<bf, st_modifier::cg> memory, const unsigned count) {
  generate_family<false>(subtree, {}, {}, oracle, memory, memory, nullptr, count);
}

EXTERN __global__ void ab_generate_memory_values_inits_and_teardowns_kernel(const __grid_constant__ ShuffleRamInitAndTeardownLayouts init_and_teardown_layouts,
                                                                            const __grid_constant__ ShuffleRamInitsAndTeardowns inits_and_teardowns,
                                                                            const matrix_setter<bf, st_modifier::cg> memory, const unsigned count) {
  generate_inits_and_teardowns<false>(init_and_teardown_layouts, inits_and_teardowns, {}, memory, memory, count);
}

EXTERN __global__ void ab_generate_memory_and_witness_values_unrolled_memory_kernel(
    const __grid_constant__ UnrolledFamilyMemorySubtree subtree,
    const __grid_constant__ OptionU32::Option<ColumnAddress> executor_family_circuit_next_timestamp_aux_var,
    const __grid_constant__ MemoryQueriesTimestampComparisonAuxVars memory_queries_timestamp_comparison_aux_vars,
    const __grid_constant__ UnrolledMemoryOracle oracle, const matrix_setter<bf, st_modifier::cg> memory, const matrix_setter<bf, st_modifier::cg> witness,
    u32 *const __restrict__ decoder_lookup_mapping, const unsigned count) {
  generate_family<true>(subtree, executor_family_circuit_next_timestamp_aux_var, memory_queries_timestamp_comparison_aux_vars, oracle, memory, witness,
                        decoder_lookup_mapping, count);
}

EXTERN __global__ void ab_generate_memory_and_witness_values_unrolled_non_memory_kernel(
    const __grid_constant__ UnrolledFamilyMemorySubtree subtree,
    const __grid_constant__ OptionU32::Option<ColumnAddress> executor_family_circuit_next_timestamp_aux_var,
    const __grid_constant__ MemoryQueriesTimestampComparisonAuxVars memory_queries_timestamp_comparison_aux_vars,
    const __grid_constant__ UnrolledNonMemoryOracle oracle, const matrix_setter<bf, st_modifier::cg> memory, const matrix_setter<bf, st_modifier::cg> witness,
    u32 *const __restrict__ decoder_lookup_mapping, const unsigned count) {
  generate_family<true>(subtree, executor_family_circuit_next_timestamp_aux_var, memory_queries_timestamp_comparison_aux_vars, oracle, memory, witness,
                        decoder_lookup_mapping, count);
}

EXTERN __global__ void ab_generate_memory_and_witness_values_inits_and_teardowns_kernel(
    const __grid_constant__ ShuffleRamInitAndTeardownLayouts init_and_teardown_layouts, const __grid_constant__ ShuffleRamInitsAndTeardowns inits_and_teardowns,
    const __grid_constant__ ShuffleRamAuxComparisonSets aux_comparison_sets, const matrix_setter<bf, st_modifier::cg> memory,
    const matrix_setter<bf, st_modifier::cg> witness, const unsigned count) {
  generate_inits_and_teardowns<true>(init_and_teardown_layouts, inits_and_teardowns, aux_comparison_sets, memory, witness, count);
}

} // namespace airbender::witness::memory::unrolled